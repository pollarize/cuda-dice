#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h> 
#include <string>
#include <time.h>
#include <chrono>     
#include <thread>         
#include <fstream>
using namespace std;

//###############################################################################################################################
// Local Defines
//###############################################################################################################################

#ifndef OPTIMIZED
	#define OPTIMIZEDOff	
#endif

#define cNumberOfBlocks           (1536)
#define cNumberOfThreadsPerBlock  (256)
#define cSizeOfDataPerThread      (128)
#define cNumberOfThreads          (cNumberOfBlocks*cNumberOfThreadsPerBlock)

#define cOutputFile               ("cudaUnit.json")

#ifndef STATUS
#define CUDA_E_OK                 ((uint8_t)0)
#define CUDA_NOT_OK               ((uint8_t)1)
#define CUDA_PENDING_OK           ((uint8_t)2)
#endif


#define mPRINT_TIME(func)                                                        \
	auto startTimer = chrono::steady_clock::now();                               \
	func;                                                                        \
	auto endTimer = chrono::steady_clock::now();                                 \
	cout << "Elapsed time in milliseconds : "                                    \
	<< chrono::duration_cast<chrono::milliseconds>(endTimer - startTimer).count()\
	<< " ms" << endl;

//###############################################################################################################################
// External Libs for CUDA 
//###############################################################################################################################
/*User libraries for CUDA*/
#include "DiceCalcCudaTypes.cuh"
#include "sha3Cuda.cuh"
#include "SwatchCuda.cuh"
#include "ValidationCuda.cuh"
#include "RandomGenCuda.cuh"
#include "DiceCudaCalculation.cuh"
#include "FIleWorker.h"

//###############################################################################################################################
// Local Types
//###############################################################################################################################

typedef enum ProgramStates {

	//Prepare Program execution
	eProgram_Init,
	eProgram_Get_Console_Options,
	eProgram_CUDA_Allocate_Memory,
	eProgram_CUDA_Cpy_Host_Memory,
	eProgram_CUDA_CURAND_Init,

	//Loop states
	eProgram_Loop_CUDA_Fill_Random,
	eProgram_Loop_CUDA_SHA3_Random,
	eProgram_Loop_Host_Time,
	eProgram_Loop_CUDA_SHA3_DICE_Proto,
	eProgram_Loop_CUDA_Validate,
	eProgram_Loop_Host_Validate,
	eProgram_Loop_Host_Display_Speed,

	//Prepare to exit
	eProgram_CUDA_Cpy_Device_Memory,
	eProgram_Host_Prepare_Check_Unit,
	eProgram_CUDA_Clean_Device_Memory,
	eProgram_Exit,

	eProgram_Count,
}EprogramStates_t;

//###############################################################################################################################
// Local Function Protorypes
//###############################################################################################################################

static void DisplayHeader(void);
static int writeToFile(diceUnitHex_t* diceUnitP);

//###############################################################################################################################
// Local Data
//###############################################################################################################################

//Host-CPU
static hipDeviceProp_t props;
static hipError_t cudaStatus;
static EprogramStates_t PStates = eProgram_Init;
static bool bIsProgramRunning = true;
static uint8_t aU8Time[sizeof(uint32_t)];
auto startTimer = chrono::steady_clock::now();
auto endTimer = chrono::steady_clock::now();
static size_t sValidDiceUnitIdx = 0;
static diceUnitHex_t diceUnitValid;
static char stringBufferL[1024];

//Device-GPU
static payload_t* pD_Payloads = 0;
static uint8_t* pD_U8Time = 0;
static diceProtoHEX_t* pD_Protos = 0;
static hashProtoHex_t* pD_ProtosShaHex = 0;
static bool* pD_ValidatingRes = 0;
static uint16_t* pD_U16Zeroes = 0;

//Copy on Host
#ifndef OPTIMIZED
static payload_t h_Payloads[cNumberOfThreads];
static hashProtoHex_t h_ProtosShaHex[cNumberOfThreads];
#endif // !OPTIMIZED

static diceProtoHEX_t h_Protos[cNumberOfThreads];
static bool h_ValidatingRes[cNumberOfThreads];
static uint16_t h_U16Zeroes;

//###############################################################################################################################
// Local Functions
//###############################################################################################################################

int main(int argc, char* argv[])
{
	hipGetDeviceProperties(&props, 0);
	hipSetDeviceFlags(hipDeviceScheduleYield | hipDeviceMapHost | hipDeviceLmemResizeToMax);

	//Stub for constant input from console
	const char* addrOp = "03037a1e2905d3bf34b31f61efcb0960ef512809";
	const char* addrMin = "0204c09f6117454ab573bd166fbef7c1e4832c1f";
	const char* zeroes = "12";

	//Set default Proto
	diceProtoHEX_t diceProtoL;
	payload_t buf_PayloadL;
	int bIsEqualL = CUDA_NOT_OK;
	uint8_t aShaReturnL[cDICE_SHA3_512_SIZE];
	uint8_t aShaHexReturnL[cDICE_UNIT_SIZE];

	//Get zeroes from string
	uint8_t aZerosL[cDICE_ZEROES_SIZE];
	hexstr_to_char((uint8_t*)zeroes, aZerosL, cDICE_ZEROES_SIZE);

	//Show data from GPU on console
	DisplayHeader();

	while (bIsProgramRunning)
	{
		switch (PStates)
		{
		case eProgram_Init:
			hipDeviceReset();
			hipDeviceReset();
			//Set current GPU Card (zero by Default for single GPU on system)
			cudaStatus = hipSetDevice(0);

			//Check for Errors
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				PStates = eProgram_Get_Console_Options;
			}
			break;

		case eProgram_Get_Console_Options:
			PStates = eProgram_CUDA_Allocate_Memory;
			break;

		case eProgram_CUDA_Allocate_Memory:
			//Allocate memory on GPU
			cudaStatus = hipMalloc((void**)&pD_Payloads, cNumberOfThreads * sizeof(payload_t));
			cudaStatus = hipMalloc((void**)&pD_Protos, cNumberOfThreads * sizeof(diceProtoHEX_t));
			cudaStatus = hipMalloc((void**)&pD_U8Time, sizeof(uint32_t));
			cudaStatus = hipMalloc((void**)&pD_ProtosShaHex, cNumberOfThreads * sizeof(hashProtoHex_t));
			cudaStatus = hipMalloc((void**)&pD_ValidatingRes, cNumberOfThreads * sizeof(bool));
			cudaStatus = hipMalloc((void**)&pD_U16Zeroes, sizeof(uint16_t));

			//Check for Errors
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMalloc failed on Payload!");
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				PStates = eProgram_CUDA_Cpy_Host_Memory;
			}
			break;

		case eProgram_CUDA_Cpy_Host_Memory:
			//Get seed Time
			getBeats(aU8Time);

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(pD_U8Time, aU8Time, sizeof(uint32_t), hipMemcpyHostToDevice);
			cudaStatus = hipMemcpy(pD_U16Zeroes, aZerosL, sizeof(uint8_t), hipMemcpyHostToDevice);

			//Set const value
			memcpy(diceProtoL.addrMin, addrMin, cDICE_ADDR_SIZE * cBYTE_TO_HEX);
			memcpy(diceProtoL.addrOp, addrOp, cDICE_ADDR_SIZE * cBYTE_TO_HEX);
			memcpy(diceProtoL.validZeroes, zeroes, cDICE_ZEROES_SIZE * cBYTE_TO_HEX);
			memset(diceProtoL.swatchTime, 1, cDICE_SWATCH_TIME_SIZE * cBYTE_TO_HEX);
			memset(diceProtoL.shaPayload, 1, cDICE_SHA3_512_SIZE * cBYTE_TO_HEX);

			//Init data in GPU with one CPU
			for (size_t i = 0; i < cNumberOfThreads; i++)
			{
				memcpy(&h_Protos[i], &diceProtoL, cDICE_PROTO_SIZE * cBYTE_TO_HEX);
			}

			cudaStatus = hipMemcpy(pD_Protos, &h_Protos, sizeof(h_Protos), hipMemcpyHostToDevice);

			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy failed!");
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				PStates = eProgram_CUDA_CURAND_Init;
			}
			break;

		case eProgram_CUDA_CURAND_Init:
			// Launch a kernel on the GPU with one thread for each element.
			gCUDA_CURAND_Init << < cNumberOfBlocks, cNumberOfThreadsPerBlock, cSizeOfDataPerThread >> > (pD_U8Time);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					PStates = eProgram_CUDA_Clean_Device_Memory;
				}
				else
				{
					PStates = eProgram_Loop_CUDA_Fill_Random;
				}
			}
			break;

			//Loop states
		case eProgram_Loop_CUDA_Fill_Random:

			// Launch a kernel on the GPU with one thread for each element.
			gCUDA_Fill_Payload << < cNumberOfBlocks, cNumberOfThreadsPerBlock, cSizeOfDataPerThread >> > (pD_Payloads);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					PStates = eProgram_CUDA_Clean_Device_Memory;
				}
				else
				{
#ifndef OPTIMIZED
					// Copy output vector from GPU buffer to host memory.
					cudaStatus = hipMemcpy(h_Payloads, pD_Payloads, cNumberOfThreads * sizeof(payload_t), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						PStates = eProgram_CUDA_Clean_Device_Memory;
					}
#endif// !OPTIMIZED
					PStates = eProgram_Loop_CUDA_SHA3_Random;
				}
			}
			break;

		case eProgram_Loop_CUDA_SHA3_Random:
			// Launch a kernel on the GPU with one thread for each element.
			gCUDA_SHA3_Random << < cNumberOfBlocks, cNumberOfThreadsPerBlock >> > (pD_Payloads, pD_Protos);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					PStates = eProgram_CUDA_Clean_Device_Memory;
				}
				else
				{
#ifndef OPTIMIZED
					// Copy output vector from GPU buffer to host memory.
					cudaStatus = hipMemcpy(h_Protos, pD_Protos, cNumberOfThreads * sizeof(diceProtoHEX_t), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						PStates = eProgram_CUDA_Clean_Device_Memory;
					}
#endif// !OPTIMIZED
					PStates = eProgram_Loop_Host_Time;
				}
			}
			break;

		case eProgram_Loop_Host_Time:
			//Get seed Time
			getBeats(aU8Time);

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(pD_U8Time, aU8Time, sizeof(uint32_t), hipMemcpyHostToDevice);

			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy failed!");
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				PStates = eProgram_Loop_CUDA_SHA3_DICE_Proto;
			}

			break;

		case eProgram_Loop_CUDA_SHA3_DICE_Proto:
			//Launch a kernel on the GPU with one thread for each element.
			gCUDA_SHA3_Proto << < cNumberOfBlocks, cNumberOfThreadsPerBlock, cSizeOfDataPerThread >> > (pD_Protos, pD_U8Time, pD_ProtosShaHex);

			//Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				//hipDeviceSynchronize waits for the kernel to finish, and returns
				//any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					PStates = eProgram_CUDA_Clean_Device_Memory;
				}
				else
				{
					//Copy output vector from GPU buffer to host memory.
#ifndef OPTIMIZED
					cudaStatus = hipMemcpy(h_Protos, pD_Protos, cNumberOfThreads * sizeof(diceProtoHEX_t), hipMemcpyDeviceToHost);
					cudaStatus = hipMemcpy(h_ProtosShaHex, pD_ProtosShaHex, cNumberOfThreads * sizeof(hashProtoHex_t), hipMemcpyDeviceToHost);

					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						PStates = eProgram_CUDA_Clean_Device_Memory;
					}
#endif// !OPTIMIZED
					PStates = eProgram_Loop_CUDA_Validate;
				}
			}
			break;

		case eProgram_Loop_CUDA_Validate:
			//Launch a kernel on the GPU with one thread for each element.
			gCUDA_ValidateProtoHash << < cNumberOfBlocks, cNumberOfThreadsPerBlock, cSizeOfDataPerThread >> > (pD_ProtosShaHex, pD_U16Zeroes, pD_ValidatingRes);

			//Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				//hipDeviceSynchronize waits for the kernel to finish, and returns
				//any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					PStates = eProgram_CUDA_Clean_Device_Memory;
				}
				else
				{
					//Copy output vector from GPU buffer to host memory.
					cudaStatus = hipMemcpy(h_ValidatingRes, pD_ValidatingRes, cNumberOfThreads * sizeof(bool), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						PStates = eProgram_CUDA_Clean_Device_Memory;
					}
					PStates = eProgram_Loop_Host_Validate;
				}
			}
			break;

		case eProgram_Loop_Host_Validate:
			PStates = eProgram_Loop_CUDA_Fill_Random;
			for (size_t i = 0; i < cNumberOfThreads; i++)
			{
				if (false == h_ValidatingRes[i])
				{
					sValidDiceUnitIdx = i;
					PStates = eProgram_CUDA_Cpy_Device_Memory;
					break;
				}
			}
			break;

			//Prepare to exit
		case eProgram_CUDA_Cpy_Device_Memory:
			cudaStatus = hipMemcpy(buf_PayloadL.payload, pD_Payloads[sValidDiceUnitIdx].payload, sizeof(payload_t), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) 
			{
				fprintf(stderr, "hipMemcpy failed!");
				PStates = eProgram_CUDA_Clean_Device_Memory;
			}
			else
			{
				PStates = eProgram_Host_Prepare_Check_Unit;
			}
			break;

		case eProgram_Host_Prepare_Check_Unit:
			//Set Up Dice Unit
			memcpy(diceUnitValid.addrOp, diceProtoL.addrOp, cDICE_ADDR_SIZE * cBYTE_TO_HEX);
			memcpy(diceUnitValid.addrMin, diceProtoL.addrMin, cDICE_ADDR_SIZE * cBYTE_TO_HEX);
			memcpy(diceUnitValid.validZeroes, diceProtoL.validZeroes, cDICE_ZEROES_SIZE * cBYTE_TO_HEX);
			dCUDA_Char_To_HexStr(aU8Time, 4, diceUnitValid.swatchTime);
			dCUDA_Char_To_HexStr(buf_PayloadL.payload, sizeof(payload_t), diceUnitValid.payload);

			//Free up GPU Memory
			PStates = eProgram_CUDA_Clean_Device_Memory;

#ifndef OPTIMIZED			
			//Check Hash of proto is as expected
			//Hash Random
			sha3_SingleExeuction(diceUnitValid.payload, cDICE_PAYLOAD_SIZE * cBYTE_TO_HEX, aShaReturnL);

			//Save data to Global Memory in HexString
			dCUDA_Char_To_HexStr(aShaReturnL, cDICE_SHA3_512_SIZE, diceProtoL.shaPayload);

			//Set Time in Global Memory for each Proto
			memcpy(diceProtoL.swatchTime, diceUnitValid.swatchTime, cDICE_SWATCH_TIME_SIZE * cBYTE_TO_HEX);

			//Hash Random
			sha3_SingleExeuction(&diceProtoL, cDICE_PROTO_SIZE * cBYTE_TO_HEX, aShaReturnL);

			//Save data to Global Memory in HexString 
			dCUDA_Char_To_HexStr(aShaReturnL, cDICE_SHA3_512_SIZE, aShaHexReturnL);

			//Check is the hash value is as expected
			bIsEqualL = memcmp(aShaHexReturnL, h_ProtosShaHex[sValidDiceUnitIdx].hashProto, cDICE_UNIT_SIZE);

			if (CUDA_E_OK != bIsEqualL)
			{
				PStates = eProgram_Loop_CUDA_Fill_Random;
			}
#endif// !OPTIMIZED
			break;

		case eProgram_CUDA_Clean_Device_Memory:
			hipFree(pD_Payloads);
			hipFree(pD_Protos);
			hipFree(pD_U8Time);
			hipFree(pD_ProtosShaHex);
			hipFree(pD_ValidatingRes);
			fprintf(stderr, "Free GPU Memory\n");
			PStates = eProgram_Exit;
			break;

		case eProgram_Exit:
#ifndef OPTIMIZED
			uint8_t aPrintReadyL[cDICE_UNIT_SIZE+1];
			memcpy(aPrintReadyL, h_ProtosShaHex[sValidDiceUnitIdx].hashProto, cDICE_UNIT_SIZE);
			aPrintReadyL[cDICE_UNIT_SIZE] = '\0';

			printf("%s\n", aPrintReadyL);
#endif // !OPTIMIZED

			writeToFile(&diceUnitValid);

			bIsProgramRunning = false;
			break;

		default:
			bIsProgramRunning = false;
			fprintf(stderr, "INVALID Program State !!!\n");
			break;
		}
	}


	return 0;
}

//###############################################################################################################################
// CPU - HOST - Functions
//###############################################################################################################################

static void DisplayHeader()
{
	const int kb = 1024;
	const int mb = kb * kb;
	wcout << "NBody.GPU" << endl << "=========" << endl << endl;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;
	//wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << endl << endl;


	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Multiprocessors: " << props.multiProcessorCount << endl;
		wcout << "  Threads per multiprocessor: " << props.maxThreadsPerMultiProcessor << endl;
		wcout << "  Concurent kernels: " << props.concurrentKernels << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << endl;
	}

}

static int writeToFile(diceUnitHex_t* diceUnitP)
{
	int iLenghtL;
	char addrOp[cDICE_ADDR_SIZE*cBYTE_TO_HEX + 1];
	char addrMin[cDICE_ADDR_SIZE*cBYTE_TO_HEX + 1];
	char zeroes[cDICE_ZEROES_SIZE*cBYTE_TO_HEX + 1];
	char swatchTime[cDICE_SWATCH_TIME_SIZE*cBYTE_TO_HEX + 1];
	char payload[cDICE_PAYLOAD_SIZE*cBYTE_TO_HEX + 1];

	memcpy(addrOp, diceUnitP->addrOp, cDICE_ADDR_SIZE*cBYTE_TO_HEX);
	memcpy(addrMin, diceUnitP->addrMin, cDICE_ADDR_SIZE*cBYTE_TO_HEX);
	memcpy(zeroes, diceUnitP->validZeroes, cDICE_ZEROES_SIZE*cBYTE_TO_HEX);
	memcpy(swatchTime, diceUnitP->swatchTime, cDICE_SWATCH_TIME_SIZE*cBYTE_TO_HEX);
	memcpy(payload, diceUnitP->payload, cDICE_PAYLOAD_SIZE*cBYTE_TO_HEX);

	addrOp[cDICE_ADDR_SIZE*cBYTE_TO_HEX] = '\0';
	addrMin[cDICE_ADDR_SIZE*cBYTE_TO_HEX] = '\0';
	zeroes[cDICE_ZEROES_SIZE*cBYTE_TO_HEX] = '\0';
	swatchTime[cDICE_SWATCH_TIME_SIZE*cBYTE_TO_HEX] = '\0';
	payload[cDICE_PAYLOAD_SIZE*cBYTE_TO_HEX] = '\0';

	iLenghtL = sprintf(stringBufferL, "\{\"addrOperator\": \"%s\",\"addrMiner\" : \"%s\",\"validZeros\" : \"%s\",\"swatchTime\" : \"%s\",	\"payLoad\" : \"%s\" \}", addrOp, addrMin, zeroes, swatchTime, payload);
	
	ofstream myfile;
	myfile.open(cOutputFile);
	myfile.write(stringBufferL,iLenghtL);
	myfile.close();

	delete[] addrOp, addrMin, zeroes, swatchTime, payload;

	return 0;
}

//###############################################################################################################################
// GPU - DEVICE - Functions
//###############################################################################################################################

